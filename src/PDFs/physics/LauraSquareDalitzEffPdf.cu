#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/DalitzPlotHelpers.h>
#include <goofit/PDFs/physics/LauraSquareDalitzEffPdf.h>
#include <vector>
#include <math.h>

namespace GooFit {

__device__ auto thetaprime( fptype s12, fptype s13) -> fptype {  //s_ij, s_ik : i = 1, j = 2, k = 3
    fptype m1 = 0.13957061;
    fptype m2=0.13957061; 
    fptype m3=0.497614;
    fptype m123=1.86483;
    fptype m12 = sqrt(s12);
    fptype e1cm12 = ((m12*m12) - (m2*m2) + (m1*m1))/(2*m12); //centre of mass energy
    fptype e3cm12 = ((m123*m123) - (m12*m12) + (m3*m3))/(2*m12);
    fptype qi = sqrt((e1cm12*e1cm12) - (m1*m1));
    fptype qk = sqrt((e3cm12*e3cm12) - (m3*m3));
    fptype cos = (s13 - (m1*m1) - (m3*m3) - (2.0*e1cm12*e3cm12))/(2.0*qi*qk);
    if (cos > 1.0){cos = 1.0;} //force within physical limit. discontinuity handled by Jaccobian
    if (cos < -1.0){cos = -1.0;}
    return acos(cos)/M_PI;
    }

__device__ auto  mprime(fptype s12) -> fptype {
    /*
    calculate m' for square dalitz formalism. 
    :param s12: mass s12
    :returns: m'
    */
    fptype m12 = sqrt(s12);
    fptype delta_m12 = 1.367216 - 0.27914122; //hardcoded for Kpi
    fptype a = (m12 - 0.27914122)/delta_m12;
    return (1/M_PI)*acos((2*a) - 1);
    }
   
__device__ auto device_LauraSquareDalitzEff(fptype *evt, ParameterContainer &pc) -> fptype {
    // Define observables
    int idx = pc.getObservable(0);
    int idy = pc.getObservable(1);

    // don't use RO_CACHE as this is used as efficiency for Amp3Body
    fptype x = evt[idx];
    fptype y = evt[idy];

    // Define coefficients
    fptype c0 = pc.getParameter(0);
    fptype c1 = pc.getParameter(1);
    fptype c2 = pc.getParameter(2);
    fptype c3 = pc.getParameter(3);
    fptype c4 = pc.getParameter(4);
    fptype c5 = pc.getParameter(5);
    fptype c6 = pc.getParameter(6);
    fptype c7 = pc.getParameter(7);
    fptype c8 = pc.getParameter(8);
    fptype c9 = pc.getParameter(9);
    fptype c10 = pc.getParameter(10);
    fptype c11 = pc.getParameter(11);
    fptype c12 = pc.getParameter(12);
    fptype c13 = pc.getParameter(13);


    fptype mD   = 1.86483;
    fptype mKS0 = 0.497611;
    fptype mh1  = 0.13957;
    fptype mh2  = 0.13957;

    pc.incrementIndex(1, 8, 0, 2, 1);

    // Check phase space
    if(!inDalitz(x, y, mD, mKS0, mh1, mh2))
        return 0;

    // Call helper functions
    fptype tp = thetaprime(x, y);
    if(tp > 1. || tp < 0.)
        return 0;
    
    fptype mp = mprime(x);
    if(mp > 1. || mp < 0.)
        return 0;

    fptype ret = c0*mp + c1*tp +c2*mp*tp + c3*mp*mp + c4*tp*tp + c5*mp*tp*tp + c6*tp*mp*mp 
               + c7*tp*tp*tp + c8*mp*mp*mp + c9*mp*mp*tp*tp + c10*mp*tp*tp*tp 
               + c11*tp*tp*tp*tp + c12*mp*mp*mp*mp +c13*mp*mp*mp*tp;

    return ret;
}

__device__ device_function_ptr ptr_to_LauraSquareDalitzEff = device_LauraSquareDalitzEff;

__host__ LauraSquareDalitzEffPdf::LauraSquareDalitzEffPdf(std::string n,
                                                Observable m12,
                                                Observable m13,
                                                Variable c0,
                                                Variable c1,
                                                Variable c2,
                                                Variable c3,
                                                Variable c4,
                                                Variable c5,
                                                Variable c6,
                                                Variable c7,
                                                Variable c8,
                                                Variable c9,
                                                Variable c10,
                                                Variable c11,
                                                Variable c12,
                                                Variable c13)

    : GooPdf("LauraSquareDalitzEffPdf", n, m12, m13, c0, c1, c2, c3, c4, c5, c6, c7, c8, c9, c10, c11, c12, c13) {
    registerFunction("ptr_to_LauraSquareDalitzEff", ptr_to_LauraSquareDalitzEff);

    initialize();
}

// __host__ fptype LauraSquareDalitzEffPdf::normalize() { return 1; }

} // namespace GooFit
